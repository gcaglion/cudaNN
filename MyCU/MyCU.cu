#include "hip/hip_runtime.h"
#include "MyCU.h"

void swap(int* v1, int* v2) {
	int tmp=(*v1);
	(*v1)=(*v2);
	(*v2)=tmp;
}

EXPORT int initCUDA() {
	// init CUDA GPU
	if (hipSetDevice(0)!=hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		return -1;
	}
	return 0;
}
EXPORT int initCUBLAS(void* cublasH) {
	// init CUBLAS

	if (hipblasCreate((hipblasHandle_t*)cublasH)!=HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization error!\n");
		return -1;
	}

	return 0;
}
EXPORT int initCURand(void* cuRandH) {
	if (hiprandCreateGenerator((hiprandGenerator_t*)cuRandH, HIPRAND_RNG_PSEUDO_DEFAULT)!=HIPRAND_STATUS_SUCCESS) {
		//if (hiprandCreateGenerator((hiprandGenerator_t*)cuRandH, HIPRAND_RNG_PSEUDO_DEFAULT)!=HIPRAND_STATUS_SUCCESS) {
		printf("CURAND initialization error!\n");
		return -1;
	}
	/* Set seed */
	if (hiprandSetPseudoRandomGeneratorSeed((*(hiprandGenerator_t*)cuRandH), 1234ULL)!=HIPRAND_STATUS_SUCCESS) return -1;
	return 0;
}
EXPORT int initCUstreams(void* cuStream[]) {
	for (int s=0; s<MAX_STREAMS; s++) {
		if (hipStreamCreate((hipStream_t*)cuStream[s])!=hipSuccess) return -1;
	}
	return 0;
}

EXPORT int Malloc_cu(numtype** var, int size) {
	return ((hipMalloc(var, size*sizeof(numtype))==hipSuccess) ? 0 : -1);
}
EXPORT int Free_cu(numtype* var) {
	return (hipFree(var));
}

//-- CPU<->GPU transfer functions
EXPORT int h2d_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if(cuStream==nullptr) {
		return ((hipMemcpy(destAddr, srcAddr, size, hipMemcpyHostToDevice)==hipSuccess)?0:-1);
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyHostToDevice, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				printf("s=%d ; CUDA error %d\n", s, hipGetLastError());
				return -1;
			}
		}
		return 0;
	}
}
EXPORT int d2h_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if (cuStream==nullptr) {
		return ((hipMemcpy(destAddr, srcAddr, size, hipMemcpyDeviceToHost)==hipSuccess) ? 0 : -1);
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyDeviceToHost, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				printf("s=%d ; CUDA error %d\n", s, hipGetLastError());
				return -1;
			}
		}
		return 0;
	}
}

__global__	void initGPUData_ker(float *data, int numElements, float value) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < numElements) {
		data[tid] = value;
	}
}
EXPORT		void initGPUData(float *data, int numElements, float value) {
	dim3 gridDim;
	dim3 blockDim;

	blockDim.x = 1024;
	gridDim.x = (numElements+blockDim.x-1)/blockDim.x;

	initGPUData_ker<<< gridDim, blockDim>>> (data, numElements, value);
}

EXPORT int loadBatchData_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	int streamSize=size/sizeof(numtype)/MAX_STREAMS;
	size_t streamBytes=streamSize*sizeof(numtype);
	for (int s=0; s<MAX_STREAMS; s++) {
		int offset=s*streamSize;
		if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyHostToDevice, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
			printf("s=%d ; CUDA error %d\n", s, hipGetLastError());
			return -1;
		}
	}
	return 0;
	//return ((hipMemcpy(destAddr, srcAddr, size, hipMemcpyHostToDevice)==hipSuccess) ? 0 : -1);
}
EXPORT int dumpArray_cu(int vlen, numtype* v, const char* fname) {
	numtype* hw=(numtype*)malloc(vlen*sizeof(numtype));
	if (hipMemcpy(hw, v, vlen*sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return -1;
	FILE* f=fopen(fname, "w");
	if (f==nullptr) return -1;
	for (int i=0; i<vlen; i++) fprintf(f, "%f\n", hw[i]);
	free(hw);
	fclose(f);
	return 0;
}
EXPORT int loadArray_cu(int vlen, numtype* v, const char* fname){
	numtype fh;
	numtype* vh=(numtype*)malloc(vlen*sizeof(numtype));
	FILE* f=fopen(fname, "r");
	if (f==nullptr) return -1;
	for (int i=0; i<vlen; i++) {
		if(fscanf(f, "%f\n", &fh)==0) return -1;
		vh[i]=fh;
	}
	if (hipMemcpy(v, vh, vlen*sizeof(numtype), hipMemcpyHostToDevice)!=hipSuccess) return -1;
	fclose(f);
	free(vh);
	return 0;
}

//-- matrix functions
EXPORT int cuMtr_cublas(void* cublasH, int my, int mx, numtype* m, numtype* otm) {
	float alpha=1;
	float beta=0;
	if (hipblasSgeam((*(hipblasHandle_t*)cublasH), HIPBLAS_OP_T, HIPBLAS_OP_T, my, mx, &alpha, m, mx, &beta, m, mx, otm, my)!=HIPBLAS_STATUS_SUCCESS) return -1;
	return 0;
}

EXPORT int MbyM_cu(void* cublasH, int Ay, int Ax, numtype Ascale, bool Atr, numtype* A, int By, int Bx, numtype Bscale, bool Btr, numtype* B, numtype* C) {

	float *alpha = &Ascale;
	float *beta = &Bscale;

	hipblasOperation_t Aop=HIPBLAS_OP_N;
	hipblasOperation_t Bop=HIPBLAS_OP_N;
	int m=Bx;
	int n=Ay;
	int k=Ax;
	int ldA=Ax;
	int ldB=Bx;
	int ldC=Bx;

	numtype* vA = A;
	numtype* vB = B;

	if (Atr) {
		Aop=HIPBLAS_OP_T;
		n=Ax; k=Ay;
	}
	if (Btr) {
		Bop=HIPBLAS_OP_T;
		m=By;
		ldC=By;
	}

	if (Vinit_cu(m*n, C, 0, 0)!=0) return -1;
	if (hipblasSgemm((*(hipblasHandle_t*)cublasH), Bop, Aop, m, n, k, alpha, vB, ldB, vA, ldA, beta, C, ldC)!=HIPBLAS_STATUS_SUCCESS) return -1;

	return 0;
}

__global__ void cuSadd(const numtype* s1, const numtype* s2, numtype* ssum) {
	ssum[0]=s1[0]+s2[0];
}
__global__ void cuVscale_ker(const int vlen, numtype *v, const numtype s) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) v[tid] *= s;
}
__global__ void cuVcopy_ker(const int vlen, const numtype *v1, numtype *v2) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) v2[tid] = v1[tid];
}
__global__ void cuVminusV_ker(const int vlen, const numtype *a, const numtype sa, const numtype *b, const numtype sb, numtype* c) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) c[tid] = a[tid]*sa-b[tid]*sb;
}
__global__ void cuVplusV_ker(const int vlen, const numtype *a, const numtype sa, const numtype *b, const numtype sb, numtype* c) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) c[tid] = a[tid]*sa+b[tid]*sb;
}
__global__ void cuVsum_ker(const int vlen, const numtype *v, numtype* osum) {

	//@@ Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*CUDA_BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2*blockIdx.x * CUDA_BLOCK_SIZE;
	if (start+t < vlen)
		partialSum[t] = v[start+t];
	else
		partialSum[t] = 0;
	if (start+CUDA_BLOCK_SIZE+t < vlen)
		partialSum[CUDA_BLOCK_SIZE+t] = v[start+CUDA_BLOCK_SIZE+t];
	else
		partialSum[CUDA_BLOCK_SIZE+t] = 0;
	//@@ Traverse the reduction tree
	for (unsigned int stride = CUDA_BLOCK_SIZE; stride>=1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t+stride];
	}
	//@@ Write the computed sum of the block to the output vector at the 
	//@@ correct index
	if (t==0)
		osum[blockIdx.x] = partialSum[0];

}
__global__ void cuVssum_ker(const int vlen, const numtype *v, numtype* ossum) {

	//@@ Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*CUDA_BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2*blockIdx.x * CUDA_BLOCK_SIZE;
	if (start+t < vlen)
		partialSum[t] = v[start+t]*v[start+t];
	else
		partialSum[t] = 0;
	if (start+CUDA_BLOCK_SIZE+t < vlen)
		partialSum[CUDA_BLOCK_SIZE+t] = v[start+CUDA_BLOCK_SIZE+t]*v[start+CUDA_BLOCK_SIZE+t];
	else
		partialSum[CUDA_BLOCK_SIZE+t] = 0;
	//@@ Traverse the reduction tree
	for (unsigned int stride = CUDA_BLOCK_SIZE; stride>=1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t+stride];
	}
	//@@ Write the computed sum of the block to the output vector at the 
	//@@ correct index
	if (t==0)
		ossum[blockIdx.x] = partialSum[0];

}
__global__ void Vscale(int vlen, numtype* v, numtype scaleM, numtype scaleP) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) v[i] = scaleM*v[i]+scaleP;
}
__global__ void Vinit_ker(int vlen, numtype* v, numtype start, numtype inc) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) v[i] = start+i*inc;
}
__global__ void VbyV2V_ker(int vlen, numtype* v1, numtype* v2, numtype* ov) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) ov[i]=v1[i]*v2[i];
}

//-- scalar functions
EXPORT int Sadd_cu(numtype* s1, numtype* s2, numtype* ssum) {
	cuSadd<<< 1, 1>>>(s1, s2, ssum);
	return ((hipGetLastError()==hipSuccess) ? 0 : -1);
}

//-- vector functions;
EXPORT int Vscale_cu(int vlen, numtype* v, numtype s){
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVscale_ker<<< gridDim, blockDim>>> (vlen, v, s);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Vcopy_cu(int vlen, numtype* v1, numtype* v2) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVcopy_ker<<< gridDim, blockDim>>> (vlen, v1, v2);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Vadd_cu(int vlen, numtype* v1, numtype scale1, numtype* v2, numtype scale2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVplusV_ker<<< gridDim, blockDim>>> (vlen, v1, scale1, v2, scale2, ov);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Vdiff_cu(int vlen, numtype* v1, numtype scale1, numtype* v2, numtype scale2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVminusV_ker<<< gridDim, blockDim>>> (vlen, v1, scale1, v2, scale2, ov);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Vsum_cu(int vlen, numtype* v, numtype* ovsum, numtype* ss_d) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVsum_ker<<< gridDim, blockDim>>> (vlen, v, ss_d );

	if (hipMemcpy(ovsum, ss_d, sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return -1;

	return ((hipGetLastError()==hipSuccess) ? 0 : -1);
}

EXPORT int Vssum_cu(int vlen, numtype* v, numtype* ovssum) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVssum_ker<<< gridDim, blockDim>>> (vlen, v, ovssum);

	return ((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Vssum_cu_cublas(void* cublasH, int Vlen, numtype* V, numtype* oVssum, numtype* ss_d) {
	if (hipblasSnrm2((*(hipblasHandle_t*)cublasH), Vlen, V, 1, oVssum)!=HIPBLAS_STATUS_SUCCESS) return -1;
	(*oVssum)=(*oVssum)*(*oVssum);
	return 0;
}

EXPORT int Vnorm_cu(void* cublasH, int Vlen, numtype* V,  numtype* oVnorm, numtype* ss_d) {
	if (hipblasSnrm2((*(hipblasHandle_t*)cublasH), Vlen, V, 1, oVnorm)!=HIPBLAS_STATUS_SUCCESS) return -1;
	if (hipMemcpy(oVnorm, ss_d, sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return -1;
	return 0;
}
EXPORT int Vinit_cu(int vlen, numtype* v, numtype start, numtype inc) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	Vinit_ker<<< gridDim, blockDim>>> (vlen, v, start, inc);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int VbyV2V_cu(int vlen, numtype* v1, numtype* v2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	VbyV2V_ker<<< gridDim, blockDim>>> (vlen, v1, v2, ov);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}

EXPORT int VinitRnd_cu(int vlen, numtype* v, numtype rndmin, numtype rndmax, void* cuRandH) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	//-- Generate n floats on device, with  values between 0.0 and 1.0, where 0.0 is excluded and 1.0 is included
	if(hiprandGenerateUniform((*(hiprandGenerator_t*)cuRandH), v, vlen) !=HIPRAND_STATUS_SUCCESS) return -1;
	//-- need to scale to rndmin<->rndmax
	Vscale<<< gridDim, blockDim>>>(vlen, v, (rndmax-rndmin), rndmax-(rndmax-rndmin)*1);

	/*/-- !!!!!!!!!!!!! REMOVE !!!!!!!!!!
	numtype* hw=(numtype*)malloc(vlen*sizeof(numtype));
	if (hipMemcpy(hw, v, vlen*sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return -1;
	char* fname = "C:/temp/rndw.txt";
	FILE* f=fopen(fname, "w");
	for (int i=0; i<vlen; i++) fprintf(f, "%f\n", hw[i]);
	free(hw);
	fclose(f);
	//--
	*/
	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}

__global__ void cuTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = tanhf(in[i]);
}
__global__ void cudTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1-tanhf(in[i])*tanhf(in[i]);
}
__global__ void ORIG_cuTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) out[i] = tanhf(in[i]);
}
__global__ void ORIG_cudTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) out[i] = 1-tanhf(in[i])*tanhf(in[i]);
}
__global__ void cuExp4_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1/(1+exp(-4*in[i]));
}
__global__ void cudExp4_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 4*exp(4*in[i])/(pow(exp(4*in[i])+1, 2));
}
__global__ void cuRelu_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = ((in[i] > 0) ? 1 : 0);
}
__global__ void cudRelu_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = ((in[i] > 0) ? in[i] : 0);
}
__global__ void cuSoftPlus_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = log(1+exp(in[i]));
}
__global__ void cudSoftPlus_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1/(1+exp(-in[i]));
}

EXPORT int Tanh_cu(int vlen, numtype* in, numtype* out) {
	/*	int blockSize=64; // The launch configurator returned block size
	int minGridSize; // The minimum grid size needed to achieve the // maximum occupancy for a full device
	int gridSize; // The actual grid size needed, based on input // size
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cudTanh_ker, 0, vlen);
	// Round up according to array size
	gridSize = (vlen+blockSize-1)/blockSize;
	cudTanh_ker<<< gridSize, blockSize>>> (vlen, in, out);
	*/
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuTanh_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int dTanh_cu(int vlen, numtype* in, numtype* out) {
/*	int blockSize=64; // The launch configurator returned block size
	int minGridSize; // The minimum grid size needed to achieve the // maximum occupancy for a full device 
	int gridSize; // The actual grid size needed, based on input // size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cudTanh_ker, 0, vlen);
	// Round up according to array size 
	gridSize = (vlen+blockSize-1)/blockSize;
	cudTanh_ker<<< gridSize, blockSize>>> (vlen, in, out);
*/
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudTanh_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Exp4_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;
	cuExp4_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int dExp4_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudExp4_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int Relu_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuRelu_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int dRelu_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudRelu_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int SoftPlus_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuSoftPlus_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
EXPORT int dSoftPlus_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudSoftPlus_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess) ? 0 : -1);
}
